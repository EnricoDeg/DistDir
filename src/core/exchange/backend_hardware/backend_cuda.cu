#include "hip/hip_runtime.h"
/*
 * @file backend_cuda.cu
 *
 * @copyright Copyright (C) 2024 Enrico Degregori <enrico.degregori@gmail.com>
 *
 * @author Enrico Degregori <enrico.degregori@gmail.com>
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:

 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.

 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.

 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "src/core/exchange/backend_hardware/backend_cuda.h"

__global__ void pack_int(int *buffer, int *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		buffer[offset+id] = data[data_idx];
	}
}

__global__ void pack_float(float *buffer, float *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		buffer[offset+id] = data[data_idx];
	}
}

__global__ void pack_double(double *buffer, double *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		buffer[offset+id] = data[data_idx];
	}
}

__global__ void unpack_int(int *buffer, int *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		data[data_idx] = buffer[offset+id];
	}
}

__global__ void unpack_float(float *buffer, float *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		data[data_idx] = buffer[offset+id];
	}
}

__global__ void unpack_double(double *buffer, double *data, int *buffer_idxlist, int buffer_size, int offset) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < buffer_size) {
		int data_idx = buffer_idxlist[offset+id];
		data[data_idx] = buffer[offset+id];
	}
}

extern "C" t_kernels * new_vtable_cuda(MPI_Datatype type) {

	t_kernels * table_kernels = (t_kernels *)malloc(sizeof(t_kernels));

	/* Malloc / Free functions */
	table_kernels->allocator = allocator_cuda;
	table_kernels->deallocator = deallocator_cuda;

	if (type == MPI_INT) {

		/* Packing / Unpacking functions */
		table_kernels->pack = (kernel_func_pack)pack_cuda_int;
		table_kernels->unpack = (kernel_func_pack)unpack_cuda_int;
	} else if (type == MPI_REAL) {

		/* Packing / Unpacking functions */
		table_kernels->pack = (kernel_func_pack)pack_cuda_float;
		table_kernels->unpack = (kernel_func_pack)unpack_cuda_float;
	} else if (type == MPI_DOUBLE) {

		/* Packing / Unpacking functions */
		table_kernels->pack = (kernel_func_pack)pack_cuda_double;
		table_kernels->unpack = (kernel_func_pack)unpack_cuda_double;
	}
}

extern "C" void pack_cuda_int(int *buffer, int *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	pack_int<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (pack_int): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void pack_cuda_float(float *buffer, float *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	pack_float<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (pack_float): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void pack_cuda_double(double *buffer, double *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	pack_double<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (pack_double): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void unpack_cuda_int(int *buffer, int *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	unpack_int<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (unpack_int): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void unpack_cuda_float(float *buffer, float *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	unpack_float<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (unpack_float): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void unpack_cuda_double(double *buffer, double *data, int *buffer_idxlist, int buffer_size, int offset) {

	int thr_per_blk = 256;
	int blk_in_grid = ceil( float(buffer_size) / thr_per_blk );

	unpack_double<<< blk_in_grid, thr_per_blk >>>(buffer, data, buffer_idxlist, buffer_size, offset);

	hipError_t err = hipDeviceSynchronize();
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (unpack_double): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void* allocator_cuda(size_t buffer_size) {

	void *p;

	hipError_t err = hipMalloc(&p, buffer_size);
	if (err == hipSuccess) {
		return p;
	} else {
		fprintf(stderr, "CUDA error (hipMalloc): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void deallocator_cuda(void *buffer) {

	hipError_t err = hipFree ( buffer );
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (hipFree): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

extern "C" void memcpy_h2d(int *buffer_cuda, int *buffer_cpu, int buffer_size) {

	hipError_t err = hipMemcpy ( buffer_cuda, buffer_cpu, (size_t)buffer_size*sizeof(int), hipMemcpyHostToDevice );
	if ( err != hipSuccess ) {
		fprintf(stderr, "CUDA error (hipMemcpy): %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}